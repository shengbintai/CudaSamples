#include "filterFnc.cuh"

/*
* Summary:				���������ֵ
* @param	d_src:		����ͼ��ָ��, ָ��һ���洢�� GPU �豸�ڴ��еĵ�ͨ�� 8-bit ͼ��
* @param	d_sum:		������ָ�룬ָ��һ���洢�� GPU �豸�ڴ��е� 8-bit ����, �����洢ͼ���
* @param	height:		ͼ��߶�
* @param	width:		ͼ����
*/
void calcFltBoxAvg(const u8* d_src, u8* d_dst, int height, int width)
{
	// source image line step
	int nSrcStep = width * sizeof(u8);

	// ROI
	NppiSize oSrcSize;
	oSrcSize.height = height;
	oSrcSize.width = width;
	NppiPoint oSrcOffset = { 0, 0 };

	NppiSize oSizeROI = { oSrcSize.width , oSrcSize.height };

	//ѡ��������͵ĳߴ��С
	NppiSize oMaskSize = { 3, 3 };
	NppiPoint oAnchor = { oMaskSize.width / 2, oMaskSize.height / 2 };

	CHECKNPP(nppiFilterBoxBorder_8u_C1R(d_src, nSrcStep, oSrcSize, oSrcOffset, d_dst, nSrcStep, oSizeROI, oMaskSize, oAnchor, NPP_BORDER_REPLICATE));

}
using namespace cv;
void calcFltConv(const u8* d_src, u8* d_dst, int height, int width)
{
	// source image line step
	int nSrcStep = width * sizeof(u8);

	// ROI
	NppiSize oSrcSize;
	oSrcSize.height = height;
	oSrcSize.width = width;
	NppiPoint oSrcOffset = { 0, 0 };

	NppiSize oSizeROI = { oSrcSize.width , oSrcSize.height };

	Mat a=Mat::ones(3,3,CV_32SC1);
	int stepBytes=4*3;
	Npp32s *kernel;// = nppiMalloc_32s_C1(4, 4, &stepBytes);
	CHECKCUDA(hipMalloc((void **)&kernel, 3 *3 * sizeof(int)));
	CHECKCUDA(hipMemcpy(kernel, a.ptr<int>(0), 3 * 3 * sizeof(int),
		hipMemcpyHostToDevice));

	CHECKCUDA(hipMemcpy(a.ptr<int>(0), kernel, 3 * 3 * sizeof(int),hipMemcpyDeviceToHost));
	//ѡ��������͵ĳߴ��С
	NppiSize oKernelSize = { 3, 3 };
	NppiPoint oAnchor = { oKernelSize.width / 2, oKernelSize.height / 2 };

	CHECKNPP(nppiFilterBorder_8u_C1R(d_src, nSrcStep, oSrcSize, oSrcOffset, d_dst, nSrcStep, oSizeROI, kernel, oKernelSize, oAnchor,2, NPP_BORDER_REPLICATE));

}